#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "dropout_layer.h"
#include "hip/hip_runtime.h"
#include "utils.h"
}

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size) input[id] = (rand[id] < prob) ? 0 : input[id]*scale;
}

#ifdef Dtype
__global__ void yoloswag420blazeit360noscope_Dtype(Dtype *input, int size, float *rand, float prob)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id < size && rand[id] < prob) input[id] = 0;
}
#endif

void forward_dropout_layer_gpu(dropout_layer layer, network net)
{
    if (!net.train) return;
    int size = layer.inputs*layer.batch;
    cuda_random(layer.rand_gpu, size);
    /*
    int i;
    for(i = 0; i < size; ++i){
        layer.rand[i] = rand_uniform();
    }
    cuda_push_array(layer.rand_gpu, layer.rand, size);
    */
#ifdef Dtype
    if (net.true_q){
        yoloswag420blazeit360noscope_Dtype<<<cuda_gridsize(size), BLOCK>>>(net.input_q_gpu, size, layer.rand_gpu, layer.probability);
    }else
#endif
    {
        yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(net.input_gpu, size, layer.rand_gpu, layer.probability, layer.scale);
    }
    check_error(hipPeekAtLastError());
}

void backward_dropout_layer_gpu(dropout_layer layer, network net)
{
    if(!net.delta_gpu) return;
    int size = layer.inputs*layer.batch;

    yoloswag420blazeit360noscope<<<cuda_gridsize(size), BLOCK>>>(net.delta_gpu, size, layer.rand_gpu, layer.probability, layer.scale);
    check_error(hipPeekAtLastError());
}
